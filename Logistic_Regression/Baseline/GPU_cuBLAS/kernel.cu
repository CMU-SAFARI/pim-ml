﻿// Logistic Regression with GD GPU baseline based on cuBLAS 
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <stdint.h> 
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i)) 

#define T float

#define MAXCHAR 500

// Read training dataset from Skin_NonSkin.txt 
static int read_input_Skin(T* X, T* Y, T* W, unsigned int m_size, unsigned int n_size) {
    printf("Reading training dataset from Skin_NonSkin.csv...\n");

    FILE* fp;
    char row[MAXCHAR];
    char* token;
    unsigned int m = 0, n = 0;

    fp = fopen("/home/yuxguo/Skin_NonSkin.csv", "r"); // add file path here 
    if (fp == NULL) {
        perror("Can't open file!");
        return(-1);
    }

    while (fgets(row, MAXCHAR, fp)) {
        token = strtok(row, ",");
        n = 0;
        while (n < n_size) {//(token != NULL) {
            X[IDX2C(m, n, m_size)] = atof(token);
            token = strtok(NULL, ",");
            n++;
        }
        char temp = atoi(token);
        if (temp == 1)
            Y[m] = 1.0;
        else
            Y[m] = 0.0;
        m++;
    }
    fclose(fp);
    printf("\nSuccessfully generate input data. m = %d\n", m);
    if (m != m_size) {
        printf("Error: invalid input m_size!\n");
        return -1;
    }
    return 0; 
}

// Read training dataset from SUSY.csv 
static int read_input_SUSY(T* X, T* Y, T* W, unsigned int m_size, unsigned int n_size) {
    printf("Reading training dataset from SUSY...\n");

    FILE* fp;
    char row[MAXCHAR];
    char* token;
    unsigned int m = 0, n = 0;

    fp = fopen("/home/yuxguo/SUSY.csv", "r"); // add file path here 
    if (fp == NULL) {
        perror("Can't open file!");
        return(-1);
    }

    while (fgets(row, MAXCHAR, fp) != NULL) {//m < m_size) {
        //fgets(row, MAXCHAR, fp); 
        token = strtok(row, ",");
        Y[m] = atof(token);

        n = 0;
        token = strtok(NULL, ",");
        while (token != NULL) {
            X[IDX2C(m, n, m_size)] = atof(token);
            token = strtok(NULL, ",");
            n++;
        }
        m++;
    }

    fclose(fp);
    printf("\nSuccessfully generate input data. m = %d\n", m);
    if (m != m_size) {
        printf("Error: invalid input m_size!\n");
        return -1;
    }
    return 0; 
}

// Create synthetic input arrays in host, X is column-major 
static void read_input(T* X, T* Y, T* W, unsigned int m_size, unsigned int n_size) {
    srand(0); 
    printf("Predefined weight: ");
    for (unsigned int w = 0; w < n_size; ++w) {
        W[w] = (T)(w + 1);
        printf("%d, ", (int)W[w]);
    }
    for (unsigned int m = 0; m < m_size; ++m) {
        for (unsigned int n = 0; n < n_size; ++n) {
            X[IDX2C(m, n, m_size)] = (float) ((float)(rand()%100000) - 50000) / 10000; 
        }
    }
    for (unsigned int m = 0; m < m_size; ++m) {
        T dot_product = 0;
        for (unsigned int n = 0; n < n_size; ++n) {
            dot_product += X[IDX2C(m, n, m_size)] * W[n] + (((float)(rand()%400) - 200)/100); 
        }
        double sigmoid_temp = 1.0 / (1.0 + exp((double)(-dot_product)));
        Y[m] = sigmoid_temp >= 0.5 ? 1.0 : 0.0;
    }
    printf("\nSuccessfully generate input data.\n");
}

// error = sigmoid(error) - Y 
__global__ void SigmoidSubY(T error[], const T Y[], unsigned int m_size, unsigned int thread_num) {
    unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    //for (unsigned int i = thread_id; i < m_size, i += thread_num;) {
    if (thread_id < thread_num) {
        error[thread_id] = (1.0 / (1.0 + expf(-error[thread_id]))) - Y[thread_id];
    }
}

// Train the model at GPU 
static void GD_GPU(hipblasHandle_t handle, T* X, T* Y, T* W,
    uint32_t m_size, uint32_t n_size, uint32_t iter_time, float lr) {
    printf("-----Start traing by cuBLAS, float-----\n");

    T* error = (T*)malloc(m_size * sizeof(T));
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // init wirght with random value
    for (uint32_t n = 0; n < n_size; ++n)
        W[n] = (T)1.0;

    T temp = (lr / m_size);
    T neg_temp = -temp;
    T one = 1.0, neg_one = -1.0, zero = 0.0;

    int threadPerBlock = 256;
    int blockNumber = (m_size + threadPerBlock - 1) / threadPerBlock;
    printf("block number: %d, thread per block: %d\n", blockNumber, threadPerBlock);

    // init GPU memory 
    hipEventRecord(start, 0);
    T* X_dev, * Y_dev, * W_dev, * error_dev, * gradient_dev;

    hipMalloc((void**)&X_dev, m_size * n_size * sizeof(T));
    hipblasSetMatrix(m_size, n_size, sizeof(T), X, m_size, X_dev, m_size);

    hipMalloc((void**)&Y_dev, m_size * sizeof(T));
    hipblasSetVector(m_size, sizeof(T), Y, 1, Y_dev, 1);

    hipMalloc((void**)&W_dev, n_size * sizeof(T));
    hipblasSetVector(n_size, sizeof(T), W, 1, W_dev, 1);

    hipMalloc((void**)&error_dev, m_size * sizeof(T));
    hipMalloc((void**)&gradient_dev, n_size * sizeof(T));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop); // init data transfer time 
    printf("inti data transfer time: %.2f\n", time); 

    // Start epochs
    hipEventRecord(start, 0);
    for (uint32_t i = 0; i < iter_time; ++i) {
        // error = Xw
        hipblasSgemv(handle, HIPBLAS_OP_N,
            m_size, n_size, &one, X_dev, m_size, W_dev, 1, &zero, error_dev, 1);

        // error = sigmoid(error) - Y 
        SigmoidSubY<<<blockNumber, threadPerBlock >>> (error_dev, Y_dev, m_size, blockNumber * threadPerBlock);

        // gradient = X_trans * error (*lr/m_size) 
        // cublasSgemv(handle, CUBLAS_OP_T, m_size, n_size, &temp, X_dev, m_size, error_dev, 1, &zero, gradient_dev, 1);
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE); 
        for (uint32_t j = 0; j < n_size; ++j) {
            hipblasSdot(handle, m_size, X_dev + j*m_size, 1, error_dev, 1, gradient_dev + j); 
        }
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST); 

        // W -= gradient*lr/m_size  
        hipblasSaxpy(handle, n_size, &neg_temp, gradient_dev, 1, W_dev, 1); 

        if (i % 100 == 0)
            printf("iter %d...\n", i); 
    } // end epochs 
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop); // GPU kernel time 
    printf("kernel time: %.2f\n", time); 

    // upload W_dev to host
    hipEventRecord(start, 0); 
    hipblasGetVector(n_size, sizeof(T), W_dev, 1, W, 1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop); // final transfer time 
    printf("final data transfer time: %.2f\n", time); 

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(X_dev);
    hipFree(Y_dev);
    hipFree(W_dev);
    hipFree(error_dev);
    hipFree(gradient_dev);
} // end GD_GPU 

void compute_error_rate(const T* X, const T* Y, const T* W, int m_size, int n_size,
    const char* comment) {
    uint32_t reduction = 0;
    uint32_t sum_of_Y = 0;

    for (int m = 0; m < m_size; ++m) {
        float dot_product = 0.0;
        for (int n = 0; n < n_size; ++n) {
            dot_product += (float)X[IDX2C(m, n, m_size)] * W[n];
        }
        double sigmoid_temp = 1 / (1 + exp((double)(-dot_product)));
        int32_t predict_temp = sigmoid_temp >= 0.5 ? 1 : 0;
        if (predict_temp != (int)Y[m]) {
            reduction++;
        }
        sum_of_Y += (int32_t)Y[m];
    }
    printf("error rate on %s = %.2f%%, reduction: %d, sum_of_Y: %d\n", comment, \
        ((float)reduction / m_size) * 100, reduction, sum_of_Y);
}

int main(void) {
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    unsigned int iter_time;
    float        learning_rate;
    unsigned int m_size;
    unsigned int n_size;

    int dataset = 1;

    // printf("Select training dataset...\n");
    // printf("1 for synthetic data, 2 for SUSY, 3 for Skin Segmentation:\n"); 
    // scanf("%d", &dataset); 
    for(dataset = 1; dataset < 4; dataset++)
    {
    if (dataset == 1) { // synthetic 
        printf("Run synthetic dataset, 6291456 x 16\n"); 
        iter_time = 500; 
        learning_rate = 0.1;
        m_size = 6291456;
        n_size = 16; 
    } 
    else if (dataset == 2) { // SUSY 
        printf("Run SUSY dataset, 5000000 x 18\n");
        iter_time = 1000;
        learning_rate = 0.1;
        m_size = 5000000;
        n_size = 18;
    }
    else if (dataset == 3) { // Skin  
        printf("Run Skin Segmentation dataset, 245057 x 3\n");
        iter_time = 500;
        learning_rate = 0.0001;
        m_size = 245057;
        n_size = 3;
    }
    else {
        printf("Please choose valid number!");
        return 1; 
    }
    printf("i = %d, lr = %.4f, m = %d, n = %d\n", iter_time, learning_rate, m_size, n_size);

    int deviceID = 0;
    hipGetDevice(&deviceID);

    hipDeviceProp_t deviceProp;

    cudaStat = hipGetDeviceProperties(&deviceProp, deviceID);

    if (cudaStat != hipSuccess) {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", cudaStat,
            __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("GPU Device %d: \"%s\" with compute capability %d.%d, global memory: %d GB\n", deviceID,
        deviceProp.name, deviceProp.major, deviceProp.minor, deviceProp.totalGlobalMem / 1024 / 1024 / 1024);

    // create handle of cuBLAS library 
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    // Pointer declaration
    T* X = (T*)malloc(m_size * n_size * sizeof(T));
    T* Y = (T*)malloc(m_size * sizeof(T));
    T* W = (T*)malloc(n_size * sizeof(T));

    if (dataset == 1) { // synthetic 
        read_input(X, Y, W, m_size, n_size); 
    }
    else if (dataset == 2) { // SUSY 
        read_input_SUSY(X, Y, W, m_size, n_size); 
    }
    else if (dataset == 3) { // Skin  
        read_input_Skin(X, Y, W, m_size, n_size); 
    }

    // start training on GPU 
    GD_GPU(handle, X, Y, W, m_size, n_size, iter_time, learning_rate);

    printf("Trained weight at GPU: ");
    for (uint32_t x = 0; x < n_size; ++x) {
        printf("%.4f, ", W[x]);
    }
    printf("\n");
    compute_error_rate(X, Y, W, m_size, n_size, "GPU");
    // printf("Training time on GPU (ms) = %.2f\n", time);

    free(X);
    free(Y);
    free(W);

    printf("--------------------------\n\n");
    }

    hipblasDestroy(handle); // destroy handle 

    return EXIT_SUCCESS;
}