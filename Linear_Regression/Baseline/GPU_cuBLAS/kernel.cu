﻿// Linear Regression with GD GPU baseline based on cuBLAS 
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h> 
#include <math.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i)) 
#define T float   
#define MAXCHAR 500

// Read training dataset from SUSY.csv, X is column-major 
static int read_input_SUSY(T* X, T* Y, T* W, unsigned int m_size, unsigned int n_size) {
    printf("Reading training dataset from SUSY.csv...\n"); 

    FILE* fp;
    char row[MAXCHAR];
    char* token;
    unsigned int m = 0, n = 0;

    fp = fopen("/home/yuxguo/SUSY.csv", "r"); // add file path here 
    if (fp == NULL) {
        perror("Can't open file!");
        return(-1);
    }

    while (fgets(row, MAXCHAR, fp)) {
        token = strtok(row, ",");
        X[IDX2C(m, 0, m_size)] = atof(token); 

        token = strtok(NULL, ",");
        Y[m] = atof(token);

        n = 1; 
        token = strtok(NULL, ",");
        while (token != NULL) {
            X[IDX2C(m, n, m_size)] = atof(token);
            token = strtok(NULL, ",");
            n++;
        }
        m++;
    }
    fclose(fp);
    printf("\nSuccessfully generate input data. m = %d\n", m);
    if (m != m_size) {
        printf("Error: invalid input m_size!\n");
        return -1;
    }
    return 0; 
}

// Create input arrays in host, X is column-major 
static void read_input(T* X, T* Y, T* W, unsigned int m_size, unsigned int n_size) {
    srand(0);
    printf("Predefined weight: ");
    for (unsigned int w = 0; w < n_size; ++w) {
        W[w] = (T)(w + 1);
        printf("%d, ", (int)W[w]); 
    }
    for (unsigned int m = 0; m < m_size; ++m) {
        for (unsigned int n = 0; n < n_size; ++n) {
            X[IDX2C(m, n, m_size)] = ((float) (rand()%10000)) / 10000; 
        }
    }
    for (unsigned int m = 0; m < m_size; ++m) {
        T tmp = 0;
        for (unsigned int n = 0; n < n_size; ++n) {
            tmp += X[IDX2C(m, n, m_size)] * W[n] + ((float) (rand()%300)) / 1000; 
        }
        Y[m] = tmp;
    }
    printf("\nSuccessfully generate input data.\n");
}

// Train the model at GPU 
static void GD_GPU(hipblasHandle_t handle, T* X, T* Y, T* W, 
                   uint32_t m_size, uint32_t n_size, uint32_t iter_time, float lr) {
    printf("-----Start traing by cuBLAS, float-----\n"); 

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // init wirght with random value
    for (uint32_t n = 0; n < n_size; ++n)
        W[n] = (T)1.0;

    T temp = (lr / m_size); 
    T neg_temp = -temp; 
    T one = 1.0, neg_one = -1.0, zero = 0.0; 

    // init GPU memory 
    hipEventRecord(start, 0);
    T *X_dev, *Y_dev, *W_dev, *error_dev, *gradient_dev;

    hipMalloc((void**)&X_dev, m_size * n_size * sizeof(T));
    hipblasSetMatrix(m_size, n_size, sizeof(T), X, m_size, X_dev, m_size); 

    hipMalloc((void**)&Y_dev, m_size * sizeof(T));
    hipblasSetVector(m_size, sizeof(T), Y, 1, Y_dev, 1);

    hipMalloc((void**)&W_dev, n_size * sizeof(T)); 
    hipblasSetVector(n_size, sizeof(T), W, 1, W_dev, 1); 

    hipMalloc((void**)&error_dev, m_size * sizeof(T)); 
    hipMalloc((void**)&gradient_dev, n_size * sizeof(T)); 

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop); // init data transfer time 
    printf("inti data transfer time: %.2f\n", time); 

    // Start epochs 
    hipEventRecord(start, 0);
    for (uint32_t i = 0; i < iter_time; ++i) {
        // error = Y 
        hipblasScopy(handle, m_size, Y_dev, 1, error_dev, 1); 

        // error = (Xw - error) * lr / m_size  
        hipblasSgemv(handle, HIPBLAS_OP_N, m_size, n_size, &temp, X_dev, m_size, W_dev, 1, &neg_temp, error_dev, 1); 

        // gradient = X_trans * error
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE); 
        for (uint32_t j = 0; j < n_size; ++j) {
            hipblasSdot(handle, m_size, X_dev + j*m_size, 1, error_dev, 1, gradient_dev + j); 
        }
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST); 

        // W -= gradient 
        hipblasSaxpy(handle, n_size, &(neg_one), gradient_dev, 1, W_dev, 1); 

        if (i % 100 == 0)
            printf("iter %d...\n", i);
    } // end epochs 
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop); // GPU kernel time 
    printf("kernel time: %.2f\n", time); 

    // upload W_dev to host
    hipEventRecord(start, 0); 
    hipblasGetVector(n_size, sizeof(T), W_dev, 1, W, 1); 

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop); // final transfer time 
    printf("final data transfer time: %.2f\n", time); 

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipFree(X_dev);
    hipFree(W_dev);
    hipFree(error_dev); 
    hipFree(gradient_dev); 
} // end GD_GPU 

static void compute_mae(const T* X, const T* Y, const T* W, int m_size, int n_size, const char* comment) {
    float reduction = 0;
    float sum_of_Y = 0;
    for (int m = 0; m < m_size; ++m) {
        float dot_product = 0.0;
        for (int n = 0; n < n_size; ++n) {
            dot_product += X[IDX2C(m,n,m_size)] * W[n]; 
        }
        reduction += (float)(fabsf(Y[m] - dot_product)) / m_size;
        sum_of_Y += Y[m] / m_size; 
    }
    // float mae = (float) reduction / m_size; 
    printf("MAE on %s = %.4f, avg Y = %.4f, error rate = %.2f%%\n", comment, reduction, sum_of_Y, \
        (reduction / sum_of_Y) * 100);
}

int main(void) {
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    unsigned int iter_time;
    float        learning_rate;
    unsigned int m_size;
    unsigned int n_size;

    int dataset = 1;

    // printf("Select training dataset...\n");
    // printf("1 for synthetic data, 2 for SUSY:\n"); 
    // scanf("%d", &dataset); 
    for(dataset = 1; dataset < 3; dataset++)
    {
    if (dataset == 1) { // synthetic 
        printf("Run synthetic dataset, 6291456 x 16\n");
        iter_time = 500;
        learning_rate = 0.1;
        m_size = 6291456;
        n_size = 16;
    }
    else if (dataset == 2) { // SUSY 
        printf("Run SUSY dataset, 5000000 x 18\n");
        iter_time = 1000;
        learning_rate = 0.1;
        m_size = 5000000;
        n_size = 18;
    }
    else {
        printf("Please choose valid number!\n");
        return 1;
    }

    printf("i = %d, lr = %.4f, m = %d, n = %d\n", iter_time, learning_rate, m_size, n_size);

    int deviceID = 0;
    hipGetDevice(&deviceID);

    hipDeviceProp_t deviceProp;

    cudaStat = hipGetDeviceProperties(&deviceProp, deviceID);

    if (cudaStat != hipSuccess) {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", cudaStat,
            __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", deviceID,
        deviceProp.name, deviceProp.major, deviceProp.minor);

    // create handle of cuBLAS library 
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    // Pointer declaration
    T* X = (T*)malloc(m_size * n_size * sizeof(T));
    T* Y = (T*)malloc(m_size * sizeof(T));
    T* W = (T*)malloc(n_size * sizeof(T));

    if (dataset == 1) { // synthetic 
        read_input(X, Y, W, m_size, n_size);
    }
    else if (dataset == 2) { // SUSY 
        read_input_SUSY(X, Y, W, m_size, n_size);
    }

    GD_GPU(handle, X, Y, W, m_size, n_size, iter_time, learning_rate); 

    printf("Trained weight at GPU: ");
    for (uint32_t x = 0; x < n_size; ++x) {
        printf("%.2f, ", W[x]);
    }
    printf("\n");

    compute_mae(X, Y, W, m_size, n_size, "GPU");

    // printf("Training time on GPU (ms) = %.2f\n", time); 

    free(X); 
    free(Y); 
    free(W); 

    printf("--------------------------\n\n"); 
    } 

    hipblasDestroy(handle); // destroy handle 

    return EXIT_SUCCESS;
}